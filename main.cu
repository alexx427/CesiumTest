#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <string>

#include "thrust/device_vector.h"

using namespace std;

template<int N = 0>
__global__ void downsample2(int * __restrict sbuf, int * __restrict dbuf, int sz)
{
    extern __shared__ int buf[];
    auto *bufb = (uint8_t*)buf;

    int wl = blockDim.x * 2;

    int ib2 = blockDim.x * blockIdx.x * 2;
    int il = threadIdx.x;

    int jl2wl = threadIdx.y * wl * 2;
    int j2sz = (blockDim.y*blockIdx.y + threadIdx.y) * 2 * sz;

    // Preload source rectangle using coalesced memory access
    buf[jl2wl + il] = sbuf[j2sz + ib2 + il];
    buf[jl2wl + blockDim.x + il] = sbuf[j2sz + ib2 + blockDim.x + il];
    buf[jl2wl + wl + il] = sbuf[j2sz + sz + ib2 + il];
    buf[jl2wl + wl + blockDim.x + il] = sbuf[j2sz + sz + ib2 + blockDim.x + il];
    __syncthreads();

    auto offl = jl2wl * 4;
    int res = 0;
    // Calculate averages of byte color components and merge them into int32
    for(int k = 0; k < 4; ++k)
    {
        // For output image sizes 2x2 and 1x1, last thread in line process N components only
        if constexpr(N != 0)
        {
            if(il == blockDim.x-1 && k >= N)
                break;
        }
        auto f = il*4+k;
        auto off = offl + f/3*6 + f%3;
        res |= ((int(bufb[off]) + int(bufb[off+3]) + int(bufb[off+wl*4]) + int(bufb[off+wl*4+3])) / 4) << (k*8);
    }

    auto *dst = &dbuf[(j2sz/2 + ib2)/2 + il];
    if constexpr(N == 2)
    {
        // for output size 2x2 first thread in line writes 4 bytes and second writes 2 bytes
        if(il == 1)
            *((int16_t*)dst) = int16_t(res);
        else
            *dst = res;
    }
    else
        *dst = res;
}

void saveImage(const thrust::device_vector<int32_t> &deviceBuf, int mipmapNum, int tsize)
{
    auto tsize4 = max(1,tsize/4);
    vector<int32_t> buf(tsize4);
    thrust::copy(deviceBuf.begin(), deviceBuf.begin() + tsize4, buf.begin());

    ofstream f("data/vancouver_"s + to_string(mipmapNum) + ".data", ios::binary);
    f.write((char*)buf.data(), tsize);
}

int main()
{
    constexpr int inputImageSize = 8192;
    ifstream f("data/vancouver.data", ios::binary);
    if(!f.is_open())
    {
        cerr << "Cannot find input file!\n";
        return -1;
    }

    vector<int32_t> buf(inputImageSize * inputImageSize * 3 / 4);
    f.read((char*)buf.data(), buf.size()*4);

    thrust::device_vector<int32_t> srcBuf(buf.begin(), buf.end());
    thrust::device_vector<int32_t> dstBuf(srcBuf.size()/4);

    unsigned tsize = 0, mipmapNum = 0;
    for(unsigned i = inputImageSize/2; i > 0; i /= 2)
    {
        auto blocky = min(32u,i);
        dim3 threads {blocky >= 4 ? blocky/4*3 : blocky, blocky, 1};
        dim3 blocks {i/blocky, i/blocky, 1};

        if(i > 2)
            downsample2<<<blocks, threads, threads.x * threads.y * 4 * sizeof(int32_t)>>>(srcBuf.data().get(), dstBuf.data().get(), i*3/2);
        else if(i > 1)
            downsample2<2><<<blocks, threads, threads.x * threads.y * 4 * sizeof(int32_t)>>>(srcBuf.data().get(), dstBuf.data().get(), i*3/2);
        else
            downsample2<3><<<blocks, threads, threads.x * threads.y * 4 * sizeof(int32_t)>>>(srcBuf.data().get(), dstBuf.data().get(), i*3/2);

        // Overlap on-device new mipmap calculation with previous mipmap saving
        if(tsize != 0)
            saveImage(srcBuf, mipmapNum, tsize);

        hipDeviceSynchronize();

        srcBuf.swap(dstBuf);
        tsize = i*i*3;
        ++mipmapNum;
    }
    saveImage(srcBuf, mipmapNum, tsize);

    return 0;
}
